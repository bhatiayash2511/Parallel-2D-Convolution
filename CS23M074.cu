/**
*   CS6023: GPU Programming 
*   Assignment 2
*   
*   Please don't change any existing code in this file.
*
*   Please add necessary memory APIs for your implementation. Use cudaFree() 
*   to free up memory as soon as you're done with an allocation. 
*   This will ensure that you don't run out of memory while running
*   large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

typedef long long ll;

__global__ void fun(long int *inputMatrix,long int *filterM,long int *outputMatrix, int m, int n, int k){
    int i = blockIdx.x;
    int j = threadIdx.x;
    extern __shared__ long int filterShared[];
    long int *filter = filterShared;
    if(threadIdx.x == 0){
      for(int i = 0; i < k*k; i++) filter[i] = filterM[i];
    }
    __syncthreads();
    long int sum = 0;
    for (int p = 0; p < k; ++p) {
        for (int q = 0; q < k; ++q) {
            int inputIndex = ((i + p - k/2) * n) + (j + q - k/2);
            int filterIndex = (p * k) + q;
            if (i + p - k/2 >= 0 && i + p - k/2 < m && j + q - k/2 >= 0 && j + q - k/2 < n) {
                sum += inputMatrix[inputIndex] * filter[filterIndex];
            }
        }
    }
    outputMatrix[i*n + j] = sum;
}

int main(int argc, char** argv) {

    int m,n,k;
    cin>>m>>n>>k;


    long int* h_mat = new long int[m * n];
    long int* h_filter = new long int[k * k];

    long int* h_ans = new long int[m * n];


    for (long int i = 0; i < m * n; i++) {
        cin>>h_mat[i];
    }

    for (long int i = 0; i < k * k; i++) {
        cin>>h_filter[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    **/

    /****************************************************Start Here***********************************************************/
    
    long int *d_input, *d_filter, *d_output;

    hipMalloc(&d_input, m * n * sizeof(long int));
    hipMalloc(&d_filter, k * k * sizeof(long int));
    hipMalloc(&d_output, m * n * sizeof(long int));

    hipMemcpy(d_input, h_mat, m * n * sizeof(long int), hipMemcpyHostToDevice);
    hipMemcpy(d_filter, h_filter, k * k * sizeof(long int), hipMemcpyHostToDevice);

    auto start = std::chrono::high_resolution_clock::now();//keep it just before the kernel launch
    fun<<<m, n,k*k*sizeof(long int)>>>(d_input, d_filter, d_output, m, n, k);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();//keep it just after the kernel launch
    
    hipMemcpy(h_ans, d_output, m * n * sizeof(long int), hipMemcpyDeviceToHost);
    
    
    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */
    
    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < m; i++) {
            for (long int j = 0; j < n; j++) {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}